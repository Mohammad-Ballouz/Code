
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void MatrixMultiplication(float* A, float* B, float* C, int height, int width, int x)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < height && col < x) {
        float Pvalue = 0;
        for (int k = 0; k < width; k++) {
            Pvalue += A[row * width + k] * B[k * x + col];
        }
        C[row * x + col] = Pvalue;
    }
}

void print_matrix(float* P, int x, int y) {
    for (int i = 0; i < x; i++) {
        for (int j = 0; j < y; j++) {
            printf("%f ", P[i * y + j]);
        }
        printf("\n");
    }
}

int main() {
    int A = 1024;
    int B = 512; 
    int C = 2048;  
    float *w1 = (float*)malloc(A * B * sizeof(float));
    float *w2 = (float*)malloc(B * C * sizeof(float));
    float *w3 = (float*)malloc(A * C * sizeof(float));

    
    srand(time(NULL));
    for (int i = 0; i < A * B; i++) {
        w1[i] = rand() / (float)RAND_MAX;
    }
    for (int i = 0; i < B * C; i++) {
        w2[i] = rand() / (float)RAND_MAX;
    }

   
    float *M, *N, *P;
    hipMalloc((void**)&M, A * B * sizeof(float));
    hipMalloc((void**)&N, B * C * sizeof(float));
    hipMalloc((void**)&P, A * C * sizeof(float));

    
    hipMemcpy(M, w1, A * B * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(N, w2, B * C * sizeof(float), hipMemcpyHostToDevice);

    
    int size = 16; 
    dim3 threadsPerBlock(size, size);
    dim3 numBlocks(ceil(C / (float)size), ceil(A / (float)size));
    
    hipEvent_t start, end;
    float elapsed_time;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);

    MatrixMultiplication<<<numBlocks, threadsPerBlock>>>(M, N, P, A, B, C);

    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&elapsed_time, start, end);

    hipEventDestroy(start);
    hipEventDestroy(end);

  
    hipMemcpy(w3, P, A * C * sizeof(float), hipMemcpyDeviceToHost);

  
    printf("Elapsed time: %f ms\n", elapsed_time);

    
    free(w1);
    free(w2);
    free(w3);
    hipFree(M);
    hipFree(N);
    hipFree(P);

    return 0;
}
